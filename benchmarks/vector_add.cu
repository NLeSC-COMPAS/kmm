#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <chrono>

#include "kmm/kmm.hpp"

using real_type = float;
const unsigned int max_iterations = 10;

__global__ void initialize_range(kmm::NDRange chunk, kmm::gpu_subview_mut<real_type> output) {
    int64_t i = blockIdx.x * blockDim.x + threadIdx.x + chunk.x.begin;
    if (i >= chunk.x.end) {
        return;
    }

    output[i] = static_cast<real_type>(i);
}

__global__ void fill_range(kmm::NDRange chunk, real_type value, kmm::gpu_subview_mut<real_type> output) {
    int64_t i = blockIdx.x * blockDim.x + threadIdx.x + chunk.x.begin;
    if (i >= chunk.x.end) {
        return;
    }

    output[i] = value;
}

__global__ void vector_add(
    kmm::NDRange range,
    kmm::gpu_subview_mut<real_type> output,
    kmm::gpu_subview<real_type> left,
    kmm::gpu_subview<real_type> right
) {
    int64_t i = blockIdx.x * blockDim.x + threadIdx.x + range.x.begin;

    if (i >= range.x.end) {
        return;
    }

    output[i] = left[i] + right[i];
}

bool inner_loop(kmm::Runtime &rt, unsigned int threads, unsigned long int n, unsigned long int chunk_size, std::chrono::duration<double> &init_time, std::chrono::duration<double> &run_time) {
    using namespace kmm::placeholders;
    dim3 block_size = threads;
    auto timing_start_init = std::chrono::steady_clock::now();
    auto A = kmm::Array<real_type> {n};
    auto B = kmm::Array<real_type> {n};
    auto C = kmm::Array<real_type> {n};

    // Initialize input arrays
    rt.parallel_submit(
        kmm::Dim {n},
        kmm::ChunkPartitioner {chunk_size},
        kmm::GPUKernel(initialize_range, block_size),
        write(A(_x))
    );
    rt.parallel_submit(
        {n},
        {chunk_size},
        kmm::GPUKernel(fill_range, block_size),
        static_cast<real_type>(1.0),
        write(B(_x))
    );
    rt.synchronize();
    auto timing_stop_init = std::chrono::steady_clock::now();
    init_time += timing_stop_init - timing_start_init;

    // Benchmark
    auto timing_start = std::chrono::steady_clock::now();
    rt.parallel_submit(
        {n},
        {chunk_size},
        kmm::GPUKernel(vector_add, block_size),
        write(C(_x)),
        A(_x),
        B(_x)
    );
    rt.synchronize();
    auto timing_stop = std::chrono::steady_clock::now();
    run_time += timing_stop - timing_start;

    // Correctness check
    std::vector<real_type> result(n);
    C.copy_to(result.data(), n);
    for ( unsigned int i = 0; i < n; i++ ) {
        if (result[i] != static_cast<real_type>(i) + 1) {
            std::cerr << "Wrong result at " << i << " : " << result[i] << " != " << static_cast<real_type>(i) + 1.0 << std::endl;
            return false;
        }
    }
    return true;
}

int main(int argc, char *argv[]) {
    auto rt = kmm::make_runtime();
    bool status = false;
    unsigned long int n = 0;
    unsigned int num_threads = 0;
    unsigned int num_chunks = 0;
    double ops = max_iterations;
    double mem = 3.0 * sizeof(real_type) * max_iterations;
    std::chrono::duration<double> init_time, vector_add_time;

    if ( argc != 4 ) {
        std::cerr << "Usage: " << argv[0] << " <threads> <num_chunks> <size>" << std::endl;
        return 1;
    }
    else {
        num_threads = std::stoi(argv[1]);
        num_chunks = std::stoi(argv[2]);
        n = std::stol(argv[3]);
    }
    ops *= n;
    mem *= n;

    // Warm-up run
    status = inner_loop(rt, num_threads, n, n, init_time, vector_add_time);
    if ( !status ) {
        std::cerr << "Warm-up run failed." << std::endl;
        return 1;
    }

    init_time = std::chrono::duration<double>();
    vector_add_time = std::chrono::duration<double>();
    for ( unsigned int iteration = 0; iteration < max_iterations; ++iteration ) {
        status = inner_loop(rt, num_threads, n, n / num_chunks, init_time, vector_add_time);
        if ( !status ) {
            std::cerr << "Run with " << num_chunks << " chunks failed." << std::endl;
            return 1;
        }
    }
    std::cout << "Performance with " << num_threads << " threads, " << num_chunks << " chunks, and n = " << n << std::endl;

    std::cout << "Total time (init): " << init_time.count() << " seconds" << std::endl;
    std::cout << "Average iteration time (init): " << init_time.count() / max_iterations << " seconds" << std::endl;

    std::cout << "Total time: " << vector_add_time.count() << " seconds" << std::endl;
    std::cout << "Average iteration time: " << vector_add_time.count() / max_iterations << " seconds" << std::endl;
    std::cout << "Throughput: " << (ops / vector_add_time.count()) / 1'000'000'000.0 << " GFLOP/s" << std::endl;
    std::cout << "Memory bandwidth: " << (mem / vector_add_time.count()) / 1'000'000'000.0 << " GB/s" << std::endl;
    std::cout << std::endl;

    return 0;
}
