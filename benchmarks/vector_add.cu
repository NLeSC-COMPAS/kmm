#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>

#include "kmm/kmm.hpp"

using real_type = float;
const unsigned int max_iterations = 10;

__global__ void initialize_range(kmm::NDRange chunk, kmm::gpu_subview_mut<real_type> output) {
    int64_t i = blockIdx.x * blockDim.x + threadIdx.x + chunk.x.begin;
    if (i >= chunk.x.end) {
        return;
    }

    output[i] = static_cast<real_type>(i);
}

__global__ void fill_range(kmm::NDRange chunk, real_type value, kmm::gpu_subview_mut<real_type> output) {
    int64_t i = blockIdx.x * blockDim.x + threadIdx.x + chunk.x.begin;
    if (i >= chunk.x.end) {
        return;
    }

    output[i] = value;
}

__global__ void vector_add(
    kmm::NDRange range,
    kmm::gpu_subview_mut<real_type> output,
    kmm::gpu_subview<real_type> left,
    kmm::gpu_subview<real_type> right
) {
    int64_t i = blockIdx.x * blockDim.x + threadIdx.x + range.x.begin;

    if (i >= range.x.end) {
        return;
    }

    output[i] = left[i] + right[i];
}

bool inner_loop(kmm::Runtime &rt, int n, int chunk_size, std::chrono::duration<double> &init_time, std::chrono::duration<double> &run_time) {
    using namespace kmm::placeholders;
    dim3 block_size = 256;
    auto timing_start_init = std::chrono::steady_clock::now();
    auto A = kmm::Array<real_type> {n};
    auto B = kmm::Array<real_type> {n};
    auto C = kmm::Array<real_type> {n};

    // Initialize input arrays
    rt.parallel_submit(
        kmm::Dim {n},
        kmm::ChunkPartitioner {chunk_size},
        kmm::GPUKernel(initialize_range, block_size),
        write(A(_x))
    );
    rt.parallel_submit(
        {n},
        {chunk_size},
        kmm::GPUKernel(fill_range, block_size),
        static_cast<real_type>(1.0),
        write(B(_x))
    );
    rt.synchronize();
    auto timing_stop_init = std::chrono::steady_clock::now();
    init_time += timing_stop_init - timing_start_init;

    // Benchmark
    auto timing_start = std::chrono::steady_clock::now();
    rt.parallel_submit(
        {n},
        {chunk_size},
        kmm::GPUKernel(vector_add, block_size),
        write(C(_x)),
        A(_x),
        B(_x)
    );
    rt.synchronize();
    auto timing_stop = std::chrono::steady_clock::now();
    run_time += timing_stop - timing_start;

    // Correctness check
    std::vector<real_type> result(n);
    C.copy_to(result.data(), n);
    for (int i = 0; i < n; i++) {
        if (result[i] != static_cast<real_type>(i) + 1) {
            std::cerr << "Wrong result at " << i << " : " << result[i] << " != " << static_cast<real_type>(i) + 1 << std::endl;
            return false;
        }
    }
    return true;
}

int main() {
    auto rt = kmm::make_runtime();
    bool status = false;
    int n = 1'000'000'000;
    double ops = n * max_iterations;
    double mem = (n * 3.0 * sizeof(real_type)) * max_iterations;
    std::chrono::duration<double> init_time, vector_add_time;

    // Warm-up run
    status = inner_loop(rt, n, n, init_time, vector_add_time);
    if ( !status ) {
        std::cerr << "Warm-up run failed." << std::endl;
        return 1;
    }

    for ( int num_chunks = 1; num_chunks < 128; num_chunks *= 2 ) {
        init_time = std::chrono::duration<double>();
        vector_add_time = std::chrono::duration<double>();
        for ( unsigned int iteration = 0; iteration < max_iterations; ++iteration ) {
            status = inner_loop(rt, n, n / num_chunks, init_time, vector_add_time);
            if ( !status ) {
                std::cerr << "Run with " << num_chunks << " chunks failed." << std::endl;
                return 1;
            }
        }
        std::cout << "Performance with " << num_chunks << " chunks" << std::endl;

        std::cout << "Total time (init): " << init_time.count() << " seconds" << std::endl;
        std::cout << "Average iteration time (init): " << init_time.count() / max_iterations << " seconds" << std::endl;

        std::cout << "Total time: " << vector_add_time.count() << " seconds" << std::endl;
        std::cout << "Average iteration time: " << vector_add_time.count() / max_iterations << " seconds" << std::endl;
        std::cout << "Throughput: " << (ops / vector_add_time.count()) / 1'000'000'000.0 << " GFLOP/s" << std::endl;
        std::cout << "Memory bandwidth: " << (mem / vector_add_time.count()) / 1'000'000'000.0 << " GB/s" << std::endl;
        std::cout << std::endl;
    }

    return 0;
}
