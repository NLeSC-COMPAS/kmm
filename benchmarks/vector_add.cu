#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>

#include "kmm/api/mapper.hpp"
#include "kmm/api/runtime.hpp"

using real_type = float;
const unsigned int max_iterations = 10;

__global__ void initialize_range(kmm::NDRange chunk, kmm::gpu_subview_mut<real_type> output) {
    int64_t i = blockIdx.x * blockDim.x + threadIdx.x + chunk.x.begin;
    if (i >= chunk.x.end) {
        return;
    }

    output[i] = static_cast<real_type>(i);
}

__global__ void fill_range(kmm::NDRange chunk, real_type value, kmm::gpu_subview_mut<real_type> output) {
    int64_t i = blockIdx.x * blockDim.x + threadIdx.x + chunk.x.begin;
    if (i >= chunk.x.end) {
        return;
    }

    output[i] = value;
}

__global__ void vector_add(
    kmm::NDRange range,
    kmm::gpu_subview_mut<real_type> output,
    kmm::gpu_subview<real_type> left,
    kmm::gpu_subview<real_type> right
) {
    int64_t i = blockIdx.x * blockDim.x + threadIdx.x + range.x.begin;

    if (i >= range.x.end) {
        return;
    }

    output[i] = left[i] + right[i];
}

int main() {
    using namespace kmm::placeholders;

    auto rt = kmm::make_runtime();
    int n = 2'000'000'000;
    unsigned long long int ops = n * max_iterations;
    unsigned long long int mem = (n * 3 * sizeof(real_type)) * max_iterations;
    int chunk_size = n / 10;
    dim3 block_size = 256;
    std::chrono::duration<double> elapsed_time;

    for ( unsigned int iteration = 0; iteration < max_iterations; ++iteration ) {
        auto A = kmm::Array<real_type> {n};
        auto B = kmm::Array<real_type> {n};
        auto C = kmm::Array<real_type> {n};

        // Initialize input arrays
        rt.parallel_submit(
            kmm::Size {n},
            kmm::ChunkPartitioner {chunk_size},
            kmm::GPUKernel(initialize_range, block_size),
            write(A(_x))
        );
        rt.parallel_submit(
            {n},
            {chunk_size},
            kmm::GPUKernel(fill_range, block_size),
            float(1.0),
            write(B(_x))
        );
        rt.synchronize();
        // Benchmark

        auto timing_start = std::chrono::steady_clock::now();
        rt.parallel_submit(
            {n},
            {chunk_size},
            kmm::GPUKernel(vector_add, block_size),
            write(C(_x)),
            A(_x),
            B(_x)
        );
        rt.synchronize();
        auto timing_stop = std::chrono::steady_clock::now();
        elapsed_time += timing_stop - timing_start;

        // Correctness check
        std::vector<real_type> result(n);
        C.copy_to(result.data(), n);
        for (int i = 0; i < n; i++) {
            if (result[i] != static_cast<real_type>(i) + 1) {
                std::cerr << "Wrong result at " << i << " : " << result[i] << " != " << float(i) + 1 << std::endl;
                return 1;
            }
        }
    }

    std::cout << "Total time: " << elapsed_time.count() << " seconds" << std::endl;
    std::cout << "Average iteration time: " << elapsed_time.count() / max_iterations << " seconds" << std::endl;
    std::cout << "Throughput: " << (ops / elapsed_time.count()) / 1'000'000'000 << " GFLOP/s" << std::endl;
    std::cout << "Memory bandwidth: " << (mem / elapsed_time.count()) / 1'000'000'000 << " GB/s" << std::endl;

    return 0;
}
