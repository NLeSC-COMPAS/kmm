#include "kmm/kmm.hpp"

void fill_array(
    kmm::NDRange region,
    kmm::subview_mut<float, 2> array,
    float value
) {
    for (auto i = region.begin(0); i < region.end(0); i++) {
        for (auto j = region.begin(1); j < region.end(1); j++) {
            array[i][j] = value;
        }
    }
}

void matrix_multiply(
    kmm::DeviceContext& device,
    kmm::NDRange region,
    int n,
    int m,
    int k,
    kmm::gpu_subview_mut<float, 2> C,
    kmm::gpu_subview<float, 2> A,
    kmm::gpu_subview<float, 2> B
) {
    using kmm::checked_cast;

    float alpha = 1.0;
    float beta = 0.0;

    const float* A_ptr = A.data_at({region.begin.x, region.begin.z});
    const float* B_ptr = B.data_at({region.begin.z, region.begin.y});
    float* C_ptr = C.data_at({region.begin.x, region.begin.y});

    KMM_GPU_CHECK(hipblasGemmEx(
        device.blas(),
        HIPBLAS_OP_T,
        HIPBLAS_OP_T,
        checked_cast<int>(region.sizes().x),
        checked_cast<int>(region.sizes().y),
        checked_cast<int>(region.sizes().z),
        &alpha,
        A_ptr,
        HIP_R_32F,
        checked_cast<int>(A.stride()),
        B_ptr,
        HIP_R_32F,
        checked_cast<int>(B.stride()),
        &beta,
        C_ptr,
        HIP_R_32F,
        checked_cast<int>(C.stride()),
        HIP_R_32F,
        HIPBLAS_GEMM_DEFAULT
    ));
}

int main() {
    using namespace kmm::placeholders;
    spdlog::set_level(spdlog::level::trace);

    auto rt = kmm::make_runtime();
    int n = 5000;
    int m = 5000;
    int k = 5000;
    int chunk_size = n / 5;

    auto A = kmm::Array<float, 2>{{n, k}};
    auto B = kmm::Array<float, 2>{{k, m}};
    auto C = kmm::Array<float, 2>{{n, m}};

    rt.parallel_submit(
        {n, k},
        {chunk_size, chunk_size},
        kmm::Host(fill_array),
        write(A, access(_x, _y)),
        1.0F
    );

    rt.parallel_submit(
        {k, m},
        {chunk_size, chunk_size},
        kmm::Host(fill_array),
        write(B, access(_x, _y)),
        1.0F
    );

    for (size_t repeat = 0; repeat < 10; repeat++) {
        C.reset();

        rt.parallel_submit(
            {n, m, k},
            {chunk_size, chunk_size, chunk_size},
            kmm::GPU(matrix_multiply),
            n,
            m,
            k,
            reduce(C, kmm::ReductionOp::Sum, access(_x, _y)),
            read(A, access(_x, _z)),
            read(B, access(_z, _y))
        );

        rt.synchronize();
    }

    return EXIT_SUCCESS;
}
