#include "hip/hip_runtime.h"
#include "spdlog/spdlog.h"

#include "kmm/kmm.hpp"

__global__ void initialize_matrix_kernel(
    kmm::Bounds<2, int> chunk,
    kmm::GPUSubviewMut<float, 2> matrix
) {
    int i = blockIdx.y * blockDim.y + threadIdx.y + chunk.y.begin;
    int j = blockIdx.x * blockDim.x + threadIdx.x + chunk.x.begin;

    if (i < chunk.y.end && j < chunk.x.end) {
        matrix[i][j] = 1.0f;
    }
}

__global__ void sum_total_kernel(
    kmm::Bounds<2, int> chunk,
    kmm::GPUSubview<float, 2> matrix,
    kmm::GPUSubviewMut<float, 2> sum
) {
    int i = blockIdx.y * blockDim.y + threadIdx.y + chunk.y.begin;
    int j = blockIdx.x * blockDim.x + threadIdx.x + chunk.x.begin;

    if (i < chunk.y.end && j < chunk.x.end) {
        sum[i][j] += matrix[i][j];
    }
}

__global__ void sum_rows_kernel(
    kmm::Bounds<2, int> chunk,
    kmm::GPUSubview<float, 2> matrix,
    kmm::GPUSubviewMut<float, 2> rows_sum
) {
    int i = blockIdx.y * blockDim.y + threadIdx.y + chunk.y.begin;
    int j = blockIdx.x * blockDim.x + threadIdx.x + chunk.x.begin;

    if (i < chunk.y.end && j < chunk.x.end) {
        rows_sum[i][j] += matrix[i][j];
    }
}

__global__ void sum_cols_kernel(
    kmm::Bounds<2, int> chunk,
    kmm::GPUSubview<float, 2> matrix,
    kmm::GPUSubviewMut<float, 2> cols_sum
) {
    int i = blockIdx.y * blockDim.y + threadIdx.y + chunk.y.begin;
    int j = blockIdx.x * blockDim.x + threadIdx.x + chunk.x.begin;

    if (i < chunk.y.end && j < chunk.x.end) {
        cols_sum[j][i] += matrix[i][j];
    }
}

int main() {
    using namespace kmm::placeholders;
    spdlog::set_level(spdlog::level::trace);

    int width = 32768;
    int height = 32768;
    int chunk_width = width / 8;
    int chunk_height = height / 8;
    auto dist = kmm::TileDomain({width, height}, {chunk_width, chunk_height});

    auto rt = kmm::make_runtime();
    auto matrix = kmm::Array<float, 2> {{height, width}};

    rt.parallel_submit(
        dist,
        kmm::GPUKernel(initialize_matrix_kernel, {16, 16}),
        bounds(_x, _y),
        write(matrix[_y][_x])
    );

    rt.synchronize();

    auto total_sum = kmm::Scalar<float>();
    auto rows_sum = kmm::Array<float>(height);
    auto cols_sum = kmm::Array<float>(width);

    rt.parallel_submit(
        dist,
        kmm::GPUKernel(sum_total_kernel, {16, 16}),
        bounds(_x, _y),
        matrix[_y][_x],
        reduce(kmm::Reduction::Sum, privatize(_y, _x), total_sum)
    );

    rt.synchronize();

    rt.parallel_submit(
        dist,
        kmm::GPUKernel(sum_rows_kernel, {16, 16}),
        bounds(_x, _y),
        matrix[_y][_x],
        reduce(kmm::Reduction::Sum, privatize(_y), rows_sum[_x])
    );

    rt.synchronize();

    rt.parallel_submit(
        dist,
        kmm::GPUKernel(sum_cols_kernel, {16, 16}),
        bounds(_x, _y),
        matrix(_y, _x),
        reduce(kmm::Reduction::Sum, privatize(_x), cols_sum[_y])
    );

    rt.synchronize();

    return EXIT_SUCCESS;
}