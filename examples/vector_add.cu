#include "hip/hip_runtime.h"
#include <iostream>

#include "spdlog/spdlog.h"

#include "kmm/array.hpp"
#include "kmm/platforms/host.hpp"
#include "kmm/runtime.hpp"

#define SIZE 65536

// __global__ void vector_add(float* A, float* B, float* C, unsigned int size) {
//     unsigned int item = (blockDim.x * blockIdx.x) + threadIdx.x;

//     if (item < size) {
//         C[item] = A[item] + B[item];
//     }
// }

void initialize(float* A, float* B, unsigned int size) {
    for (unsigned int item = 0; item < size; item++) {
        reinterpret_cast<float*>(A)[item] = 1.0;
        reinterpret_cast<float*>(B)[item] = 2.0;
    }
    //    std::cout << "initialize\n";
}

void execute(float* C, const float* A, const float* B, unsigned int size) {
    for (unsigned int item = 0; item < size; item++) {
        C[item] = A[item] + B[item];
    }
    //    std::cout << "execute\n";
}

void verify(const float* C, unsigned int size) {
    for (unsigned int item = 0; item < size; item++) {
        if ((C[item] - 3.0) > 1.0e-9) {
            std::cout << "ERROR" << std::endl;
            break;
        }
    }
    //    std::cout << "SUCCESS" << std::endl;
}

int main(void) {
    spdlog::set_level(spdlog::level::warn);

    unsigned int threads_per_block = 1024 * 1024;
    unsigned int n_blocks = ceil((1.0 * SIZE) / threads_per_block);
    int n = SIZE;

    // Create manager
    auto manager = kmm::build_runtime();

    for (size_t i = 0; i < 100000; i++) {
        // Request 3 memory areas of a certain size
        auto A = kmm::Array<float>({n});
        auto B = kmm::Array<float>({n});
        auto C = kmm::Array<float>({n});

        manager.submit(kmm::Host(), initialize, write(A), write(B), 100);
        manager.submit(kmm::Host(), execute, write(C), A, B, 100);
        manager.submit(kmm::Host(), verify, C, 100);
    }

    manager.synchronize();
    std::cout << "done\n";

    return 0;
}
