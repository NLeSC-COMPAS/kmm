#include "hip/hip_runtime.h"
#include "kmm/api/mapper.hpp"
#include "kmm/api/runtime.hpp"

__global__ void initialize_range(
    kmm::WorkRange chunk,
    kmm::cuda_subview_mut<float> output
) {
    int64_t i = blockIdx.x * blockDim.x +  threadIdx.x + chunk.begin();
    if (i >= chunk.end()) {
        return;
    }

    output[i] = float(i);
}

__global__ void fill_range(
    kmm::WorkRange chunk,
    float value,
    kmm::cuda_subview_mut<float> output
) {
    int64_t i = blockIdx.x * blockDim.x +  threadIdx.x + chunk.begin();
    if (i >= chunk.end()) {
        return;
    }

    output[i] = value;
}

__global__ void vector_add(
    kmm::WorkRange chunk,
    kmm::cuda_subview_mut<float> output,
    kmm::cuda_subview<float> left,
    kmm::cuda_subview<float> right
) {
    int64_t i = blockIdx.x * blockDim.x +  threadIdx.x + chunk.begin();
    if (i >= chunk.end()) {
        return;
    }

    output[i] = left[i] + right[i];
}

int main() {
    using namespace kmm::placeholders;
    spdlog::set_level(spdlog::level::trace);

    auto rt = kmm::make_runtime();
    int n = 2'000'000'000;
    int chunk_size = n / 10;
    dim3 block_size = 256;

    auto A = kmm::Array<float>{n};
    auto B = kmm::Array<float>{n};
    auto C = kmm::Array<float>{n};

    rt.parallel_submit(
        {n},
        {chunk_size},
        kmm::CudaKernel(initialize_range, block_size),
        write(A, access(_x))
    );

    rt.parallel_submit(
        {n},
        {chunk_size},
        kmm::CudaKernel(fill_range, block_size),
        float(M_PI),
        write(B, access(_x))
    );

    rt.parallel_submit(
        {n},
        {chunk_size},
        kmm::CudaKernel(vector_add, block_size),
        write(C, access(_x)),
        read(A, access(_x)),
        read(B, access(_x))
    );

    std::vector<float> result(n);
    C.copy_to(result.data());

    return 0;
}
