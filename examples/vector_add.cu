#include "hip/hip_runtime.h"
#include <iostream>

#include "kmm/api/mapper.hpp"
#include "kmm/api/runtime.hpp"

__global__ void initialize_range(kmm::NDRange chunk, kmm::gpu_subview_mut<float> output) {
    int64_t i = blockIdx.x * blockDim.x + threadIdx.x + chunk.x.begin;
    if (i >= chunk.x.end) {
        return;
    }

    output[i] = float(i);
}

__global__ void fill_range(kmm::NDRange chunk, float value, kmm::gpu_subview_mut<float> output) {
    int64_t i = blockIdx.x * blockDim.x + threadIdx.x + chunk.x.begin;
    if (i >= chunk.x.end) {
        return;
    }

    output[i] = value;
}

__global__ void vector_add(
    kmm::NDRange range,
    kmm::gpu_subview_mut<float> output,
    kmm::gpu_subview<float> left,
    kmm::gpu_subview<float> right
) {
    int64_t i = blockIdx.x * blockDim.x + threadIdx.x + range.x.begin;

    if (i >= range.x.end) {
        return;
    }

    output[i] = left[i] + right[i];
}

int main() {
    using namespace kmm::placeholders;
    spdlog::set_level(spdlog::level::trace);

    auto rt = kmm::make_runtime();
    int n = 2'000'000'000;
    int chunk_size = n / 10;
    dim3 block_size = 256;

    auto A = kmm::Array<float> {n};
    auto B = kmm::Array<float> {n};
    auto C = kmm::Array<float> {n};

    rt.parallel_submit(
        kmm::Size {n},
        kmm::ChunkPartitioner {chunk_size},
        kmm::GPUKernel(initialize_range, block_size),
        write(A(_x))
    );

    rt.parallel_submit(
        {n},
        {chunk_size},
        kmm::GPUKernel(fill_range, block_size),
        float(1.0),
        write(B(_x))
    );

    rt.parallel_submit(
        {n},
        {chunk_size},
        kmm::GPUKernel(vector_add, block_size),
        write(C(_x)),
        A(_x),
        B(_x)
    );

    std::vector<float> result(n);
    C.copy_to(result.data(), n);
    // Correctness check
    for (int i = 0; i < n; i++) {
        if (result[i] != float(i) + 1) {
            std::cerr << "Wrong result at " << i << " : " << result[i] << " != " << float(i) + 1 << std::endl;
            return 1;
        }
    }
    std::cout << "Correctness check completed." << std::endl;

    return 0;
}
