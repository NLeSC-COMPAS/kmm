#include "hip/hip_runtime.h"
#include "vector_add.h"
#include "vector_add.cuh"


int main(void) {
    unsigned int threads_per_block = 256;
    unsigned int n_blocks = ceil((1.0 * SIZE) / threads_per_block);
    int n = SIZE;

    // Allocate memory on the host
    auto A = new float[n];
    auto B = new float[n];
    auto C = new float[n];

    // Initialize array A and B on the host
    initialize(A, B);

    // Allocate memory on the device
    void *A_d, *B_d, *C_d;
    hipMalloc(&A_d, n * sizeof(float ));
    hipMalloc(&B_d, n * sizeof(float ));
    hipMalloc(&C_d, n * sizeof(float ));

    // Copy A and B from host to device
    hipMemcpy(A_d, A, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_d, B, n * sizeof(float), hipMemcpyHostToDevice);

    // Execute the function on the device
    vector_add<<<n_blocks, threads_per_block>>>(reinterpret_cast<float *>(A_d), reinterpret_cast<float *>(B_d), reinterpret_cast<float *>(C_d), n);

    // Copy C from device to host
    hipMemcpy(C, C_d, n * sizeof(float), hipMemcpyDeviceToHost);

    // Verify the result on the host.
    hipDeviceSynchronize();
    verify(C);

    std::cout << "done\n";

    return 0;
}
