#include "kmm.hpp"

namespace kmm {

// Misc

inline void cudaErrorCheck(hipError_t err, std::string message) {
    if (err != hipSuccess) {
        throw std::runtime_error(message);
    }
}

void cudaCopyD2H(CUDA& device, Buffer& source, Buffer& target, Stream& stream) {
    auto err = hipMemcpyAsync(
        target.getPointer(),
        source.getPointer(),
        target.getSize(),
        hipMemcpyDeviceToHost,
        stream.getStream(device));
    cudaErrorCheck(err, "Impossible to copy memory from device to host.");
}

void cudaCopyD2H(CUDA& device, Buffer& source, void* target, Stream& stream) {
    auto err = hipMemcpyAsync(
        target,
        source.getPointer(),
        source.getSize(),
        hipMemcpyDeviceToHost,
        stream.getStream(device));
    cudaErrorCheck(err, "Impossible to copy memory from device to host.");
}

void cudaCopyH2D(CUDA& device, Buffer& source, Buffer& target, Stream& stream) {
    auto err = hipMemcpyAsync(
        target.getPointer(),
        source.getPointer(),
        target.getSize(),
        hipMemcpyHostToDevice,
        stream.getStream(device));
    cudaErrorCheck(err, "Impossible to copy memory from host to device.");
}

void cudaCopyD2D(CUDA& device, Buffer& source, Buffer& target, Stream& stream) {
    auto err = hipMemcpyAsync(
        target.getPointer(),
        source.getPointer(),
        target.getSize(),
        hipMemcpyDeviceToDevice,
        stream.getStream(device));
    cudaErrorCheck(err, "Impossible to copy memory from device to device.");
}

// Buffer

void Buffer::allocate(CUDA& device, Stream& stream) {
    auto err = hipMallocAsync(&(this->buffer), size, stream.getStream(device));
    cudaErrorCheck(err, "Impossible to allocate CUDA memory.");
}

void Buffer::allocate(CUDAPinned& memory) {
    auto err = hipHostMalloc(&(this->buffer), size);
    cudaErrorCheck(err, "Impossible to allocate Pinned host memory.");
}

void Buffer::destroy(CUDA& device, Stream& stream) {
    auto err = hipFreeAsync(this->buffer, stream.getStream(device));
    cudaErrorCheck(err, "Impossible to release memory.");
    this->buffer = nullptr;
    this->size = 0;
    this->device = std::make_shared<UnknownDevice>();
}

//Stream

Stream::Stream() {
    this->hip_stream = nullptr;
}

Stream::Stream(CUDA& device) {
    try {
        auto err = hipStreamCreate(&(this->hip_stream));
        cudaErrorCheck(err, "Impossible to create CUDA stream.");
    } catch (const std::exception& e) {
        std::cerr << e.what() << std::endl;
        this->hip_stream = nullptr;
    }
}

Stream::~Stream() {
    if (this->hip_stream != nullptr) {
        try {
            auto err = hipStreamDestroy(this->hip_stream);
            cudaErrorCheck(err, "Impossible to destroy CUDA stream.");
        } catch (const std::exception& e) {
            std::cerr << e.what() << std::endl;
        }
    }
}

hipStream_t Stream::getStream(CUDA& device) {
    return this->hip_stream;
}

}  // namespace kmm
