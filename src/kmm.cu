#include "kmm/kmm.hpp"

namespace kmm {

// Misc

inline void cudaErrorCheck(hipError_t err, std::string message) {
    if (err != hipSuccess) {
        throw std::runtime_error(message);
    }
}

void cudaCopyD2H(CUDA& device, Buffer& source, Buffer& target, Stream& stream) {
    auto err = hipMemcpyAsync(
        target.getPointer(),
        source.getPointer(),
        target.size(),
        hipMemcpyDeviceToHost,
        stream.getStream(device));
    cudaErrorCheck(err, "Impossible to copy memory from device to host.");
}

void cudaCopyD2H(CUDA& device, Buffer& source, void* target, Stream& stream) {
    auto err = hipMemcpyAsync(
        target,
        source.getPointer(),
        source.size(),
        hipMemcpyDeviceToHost,
        stream.getStream(device));
    cudaErrorCheck(err, "Impossible to copy memory from device to host.");
}

void cudaCopyH2D(CUDA& device, Buffer& source, Buffer& target, Stream& stream) {
    auto err = hipMemcpyAsync(
        target.getPointer(),
        source.getPointer(),
        target.size(),
        hipMemcpyHostToDevice,
        stream.getStream(device));
    cudaErrorCheck(err, "Impossible to copy memory from host to device.");
}

void cudaCopyD2D(CUDA& device, Buffer& source, Buffer& target, Stream& stream) {
    auto err = hipMemcpyAsync(
        target.getPointer(),
        source.getPointer(),
        target.size(),
        hipMemcpyDeviceToDevice,
        stream.getStream(device));
    cudaErrorCheck(err, "Impossible to copy memory from device to device.");
}

// Buffer

void Buffer::allocate(CUDA& device, Stream& stream) {
    auto err = hipMallocAsync(&(this->buffer_), size_, stream.getStream(device));
    cudaErrorCheck(err, "Impossible to allocate CUDA memory.");
}

void Buffer::allocate(CUDAPinned& memory) {
    auto err = hipHostMalloc(&(this->buffer_), size_);
    cudaErrorCheck(err, "Impossible to allocate Pinned host memory.");
}

void Buffer::destroy(CUDA& device, Stream& stream) {
    auto err = hipFreeAsync(this->buffer_, stream.getStream(device));
    cudaErrorCheck(err, "Impossible to release memory.");
    this->buffer_ = nullptr;
    this->size_ = 0;
    this->device_ = std::make_shared<UnknownDevice>();
}

//Stream

Stream::Stream() {
    this->hip_stream = nullptr;
}

Stream::Stream(CUDA& device) {
    try {
        auto err = hipStreamCreate(&(this->hip_stream));
        cudaErrorCheck(err, "Impossible to create CUDA stream.");
    } catch (const std::exception& e) {
        std::cerr << e.what() << std::endl;
        this->hip_stream = nullptr;
    }
}

Stream::~Stream() {
    if (this->hip_stream != nullptr) {
        try {
            auto err = hipStreamDestroy(this->hip_stream);
            cudaErrorCheck(err, "Impossible to destroy CUDA stream.");
        } catch (const std::exception& e) {
            std::cerr << e.what() << std::endl;
        }
    }
}

hipStream_t Stream::getStream(CUDA& device) {
    return this->hip_stream;
}

}  // namespace kmm
